#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <math.h>

#define CUB_IGNORE_DEPRECATED_CPP_DIALECT

//#ifndef __HIPCC__
#define __HIPCC__ 1
//#endif

// C:/Program Files/NVIDIA GPU Computing Toolkit/CUDA/v10.1/include/
#include <hip/hip_cooperative_groups.h>
//#include <sm_30_intrinsics.hpp>
using namespace cooperative_groups;

#include "macros.h"
#include "playground_host.h"
#include "common_funcs.cu"

#if 1
#define PRINTZ(fmt, ...) printf(fmt"\n", ##__VA_ARGS__)
#else
#define PRINTZ(fmt, ...)
#endif

  #define WIN32_LEAN_AND_MEAN
  #include <windows.h>

double PCFreq = 0.0;
__int64 timerStart = 0;

void read_values_from_file(const char * file, float * data, size_t size)
{
    std::ifstream values(file, std::ios::binary);
    if(!values.is_open())
        throw std::runtime_error("Unable to read file " + std::string(file));

    values.read(reinterpret_cast<char*>(data), size);
    values.close();
}

void write_values_to_file(const char * file, float * data, size_t size) {
    std::ofstream values(file, std::ios::binary);
    if(!values.is_open())
        throw std::runtime_error("Unable to write file " + std::string(file));

    values.write(reinterpret_cast<char*>(data), size);
    values.close();
}


void StartTimer()
{
  LARGE_INTEGER li;
  if(!QueryPerformanceFrequency(&li))
    printf("QueryPerformanceFrequency failed!\n");

  PCFreq = (double)li.QuadPart/1000.0;

  QueryPerformanceCounter(&li);
  timerStart = li.QuadPart;
}

// time elapsed in ms
double GetTimer()
{
  LARGE_INTEGER li;
  QueryPerformanceCounter(&li);
  return (double)(li.QuadPart-timerStart)/PCFreq;
}

#define SOFTENING 1e-7f

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

struct Body {
    float x, y, z, vx, vy, vz;
};

/*
 * Calculate the gravitational impact of all bodies in the system
 * on all others.
 */
template < uint32_t BlockSz >
__global__ void bodyForceKernel(Body *InOut, float dt, int n)
{
    uint32_t thid = threadIdx.x;

//    PRINTZ("%d: ii = %d", thid, ii);
//    auto res = prefixSum< BlockSz >(ii, [](uint32_t& lhs, uint32_t rhs) { lhs += rhs; });
//    PRINTZ("%d: ii = %d; res = %d", thid, ii, res);
//    return;

    __shared__ Body B[1];

  // one thread block runs for one 'i'
   // iterations over 'i' are independent
  for (int i = blockIdx.x; i < n; i += gridDim.x) {

    struct UU { float vx, vy, vz; };
    UU S{};

    if(thid == 0)
        B[0] = InOut[i];
    __syncthreads();

    // one block is responsible for inner loop
    for (int j = thid; j < n; j += BlockSz) {
    //for (int j = 0; j < n; j++) {
      float dx = InOut[j].x - InOut[i].x;
      float dy = InOut[j].y - InOut[i].y;
      float dz = InOut[j].z - InOut[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;
      //float dist = sqrtf(distSqr),
       //     invDist3 = 1.0f / (distSqr * dist);

      S.vx += dx * invDist3;
      S.vy += dy * invDist3;
      S.vz += dz * invDist3;
    }
#if 0
    __shared__ UU UUX[BlockSz];
    UUX[thid] = S;
    __syncthreads();

    if(thid == 0) {
        UU res = {};
        for(int j = 0; j < BlockSz; j++) {
            res.vx += UUX[j].vx;
            res.vy += UUX[j].vy;
            res.vz += UUX[j].vz;
        }

        InOut[i].vx = B[0].vx + dt*res.vx;
        InOut[i].vy = B[0].vy + dt*res.vy;
        InOut[i].vz = B[0].vz + dt*res.vz;
    }

#else

    auto cta = this_thread_block();
    // actually we do not need the full prefix sum but just total sum..
    auto res = prefixSum< BlockSz >(cta, S, [](UU& lhs, const UU& rhs)
            { lhs.vx += rhs.vx, lhs.vy += rhs.vy, lhs.vz += rhs.vz; }
    );

    uint32_t numThids = n % BlockSz;
    if(numThids == 0)
        numThids = BlockSz;
    //__syncthreads();
    // n is divided between several threads: in total 4096 / 128

    // the last thread holds the total sum
    if(thid == numThids-1) {
        InOut[i].vx += dt*res.vx;
        InOut[i].vy += dt*res.vy;
        InOut[i].vz += dt*res.vz;
    }
#endif
    __syncthreads();
  } // for i
}

template < uint32_t BlockSz >
__global__ void accumulateKernel(Body *p, float dt, int n)
{
    int idx = blockIdx.x*BlockSz + threadIdx.x,
        step = BlockSz * gridDim.x;
    for (int i = idx; i < n; i += step) { // integrate position
        p[i].x += p[i].vx*dt;
        p[i].y += p[i].vy*dt;
        p[i].z += p[i].vz*dt;
    }
}


#define CUDA_CHECK { \
    auto res = hipGetLastError(); \
    if(res != hipSuccess) \
       fprintf(stderr, "%d: CUDA error: %s\n", __LINE__, hipGetErrorString(res)); \
    }

// http://en.wikipedia.org/wiki/Kahan_summation_algorithm
__device__ __host__ void bodyForce(Body *p, float dt, int n)
{
  for (int i = 0; i < n; ++i) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }

    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
  }
}


static void computeCPU(const char *fname, Body *p, int nBodies, int nIters, const float dt)
{
    PRINTZ("Computing reference solution");
    int bytes = nBodies*sizeof(Body);
    read_values_from_file(fname, (float *)p, bytes);

    for (int iter = 0; iter < nIters; iter++) {
    /*
     * You will likely wish to refactor the work being done in `bodyForce`,
     * and potentially the work to integrate the positions.
     */

      bodyForce(p, dt, nBodies); // compute interbody forces

    /*
     * This position integration cannot occur until this round of `bodyForce` has completed.
     * Also, the next round of `bodyForce` cannot begin until the integration is complete.
     */

      for (int i = 0 ; i < nBodies; i++) { // integrate position
        p[i].x += p[i].vx*dt;
        p[i].y += p[i].vy*dt;
        p[i].z += p[i].vz*dt;
      }
    }
}

int nBodySim(const int argc, const char** argv)
{
    int deviceId = -1, numberOfSMs = 0;
    hipGetDevice(&deviceId);                  // deviceId: now points to the id of the currently active GPU.
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
    PRINTZ("deviceID: %d; Num SMs: %d", deviceId, numberOfSMs);

    // The assessment will test against both 2<11 and 2<15.
    // Feel free to pass the command line argument 15 when you generate ./nbody report files
    int nBodies = 2<<11;
    if (argc > 1) nBodies = 2<<atoi(argv[1]);

    // The assessment will pass hidden initialized values to check for correctness.
    // You should not make changes to these files, or else the assessment will not work.
    const char * initialized_values;
    const char * solution_values;

    if (nBodies == 2<<11) {
        initialized_values = "initialized_4096";
        solution_values = "solution_4096";
    } else { // nBodies == 2<<15
        initialized_values = "initialized_65536";
        solution_values = "solution_65536";
    }

    if (argc > 2) initialized_values = argv[2];
    if (argc > 3) solution_values = argv[3];

    const float dt = 0.01f; // Time step
    const int nIters = 10;  // Simulation iterations

    Body *p = nullptr;
    auto bytes = nBodies * sizeof(Body);
    hipMallocManaged(&p, bytes);
    //hipMemPrefetchAsync(p, bytes, hipCpuDeviceId);

    read_values_from_file(initialized_values, (float *)p, bytes);

    double totalTime = 0.0;

    /*
   *     This simulation will run for 10 cycles of time, calculating gravitational
    * interaction amongst bodies, and adjusting their positions to reflect.
    */

//  auto err = hipGetLastError();
//  PRINTZ("last error: %s", hipGetErrorString(err));

    for (int iter = 0; iter < nIters; iter++) {
        StartTimer();
        //hipMemPrefetchAsync(p, bytes, deviceId);

        /*
        * You will likely wish to refactor the work being done in `bodyForce`,
        * and potentially the work to integrate the positions.
        */

        constexpr uint32_t BlockSz = 128;
        uint32_t nBlocks = nBodies / BlockSz;
        bodyForceKernel< BlockSz ><<<nBodies, BlockSz>>>(p, dt, nBodies);

        accumulateKernel< BlockSz ><<<nBlocks, BlockSz>>>(p, dt, nBodies);

        hipDeviceSynchronize();

        //hipMemPrefetchAsync(p, bytes, hipCpuDeviceId);

        /*
        * This position integration cannot occur until this round of `bodyForce` has completed.
        * Also, the next round of `bodyForce` cannot begin until the integration is complete.
        */


        CUDA_CHECK
        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }

//  write_values_to_file(solution_values, (float *)p, bytes);
//  return 1;

  double avgTime = totalTime / (double)(nIters);
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

  std::vector< Body > truth(nBodies);
   //read_values_from_file(solution_values, (float *)p, bytes);
  computeCPU(initialized_values, truth.data(), nBodies, nIters, dt);

  for(uint32_t i = 0; i < truth.size(); i++) {

      const float eps = 1e-2;
      const Body& pT = truth[i], pC = p[i];

      auto F = [&i, &eps](auto check, auto truth) {
         auto d = std::abs(check - truth), abst = std::abs(truth);
         if(abst > 1e-3)
             d /= abst;
         if(d > eps) {
             PRINTZ("%d: %f -- %f: diff: %f", i, truth, check, d);
         }
      };

      F(pC.x, pT.x); F(pC.y, pT.y); F(pC.z, pT.z);
      F(pC.vx, pT.vx); F(pC.vy, pT.vy); F(pC.vz, pT.vz);
  }

  // You will likely enjoy watching this value grow as you accelerate the application,
  // but beware that a failure to correctly synchronize the device might result in
  // unrealistically high values.
  printf("%0.3f Billion Interactions / second\n", billionsOfOpsPerSecond);

  hipFree(p);

  return 1;
}
